#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdlib.h>
#include <omp.h>
#include<iostream>
#include <cstdlib>
#include <vector>
#include <chrono>
#include <cmath>
#include <numeric>

// N = 2^27
#define N pow(2, 27)

#define THREAD_COUNT 256
#define BLOCK_COUNT 262144

using namespace std;

__global__ void reduce4(int *g_idata, int *g_odata) {
	__shared__ int sdata[THREAD_COUNT];
	// each thread loads one element from global to shared mem
	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
	__syncthreads();
	// do reduction in shared mem
	for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	if (tid < 32)
	{
		sdata[tid] += sdata[tid + 32];
		sdata[tid] += sdata[tid + 16];
		sdata[tid] += sdata[tid + 8];
		sdata[tid] += sdata[tid + 4];
		sdata[tid] += sdata[tid + 2];
		sdata[tid] += sdata[tid + 1];
	}
	// write result for this block to global mem
	if (tid == 0) {
		g_odata[blockIdx.x] = sdata[0];
		//printf("%d\n" , g_odata[blockIdx.x]);
	}
}


int main()
{
		hipSetDevice(0);
		int* a_0 = (int*)malloc(N * sizeof(int));
		int* b_0 = (int*)malloc(BLOCK_COUNT * sizeof(int));
		int *dev_a_0 = 0;
		int *dev_b_0 = 0;
		
		int my_sum = 0;
		for (unsigned long long i = 0; i < N ;i++){
			a_0[i] = (int)rand() % 10;
			my_sum += a_0[i];
		}
		for (unsigned long long i = 0; i < BLOCK_COUNT ;i++){
			b_0[i] = 0;
		}
		
		cout<<"total sum in a: "<<my_sum<<endl;
		
		hipError_t cudaStatus;

		double start_time = omp_get_wtime();

		cudaStatus = hipMalloc((void**)&dev_a_0, N * sizeof(int));
		
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc dev_a failed!");
			hipFree(dev_a_0);
			free(a_0);
			free(b_0);
			return 1;
		}
		
		cudaStatus = hipMalloc((void**)&dev_b_0, BLOCK_COUNT * sizeof(int));
		
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc dev_a failed!");
			hipFree(dev_a_0);
			hipFree(dev_b_0);
			free(a_0);
			free(b_0);
			return 1;
		}
			
		hipMemcpy(dev_a_0, a_0, N * sizeof(int), hipMemcpyHostToDevice);
		
		double t1 = omp_get_wtime();
		reduce4 << <BLOCK_COUNT, THREAD_COUNT>> > (dev_a_0, dev_b_0);
		hipDeviceSynchronize();
		double t2 = omp_get_wtime();
		std::cout <<"computational time: "<<t2 - t1<<endl;
		
		hipMemcpy(b_0, dev_b_0, BLOCK_COUNT * sizeof(int), hipMemcpyDeviceToHost);
		
		double end_time = omp_get_wtime();
		std::cout <<"time: "<<end_time - start_time<<endl;
		
		my_sum = 0;
		for (unsigned long long i = 0; i < BLOCK_COUNT ;i++){
			my_sum += b_0[i];
		}
		
		cout<<"total sum in a (computed in GPU): "<<my_sum<<endl;
		
		hipFree(dev_a_0);
		hipFree(dev_b_0);
		free(a_0);
		free(b_0);
		
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}
		
		return 0;
}