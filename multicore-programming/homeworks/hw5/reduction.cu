/*
* In his exalted name
*
* Reduction - Sequential Code
* Written by Ahmad Siavashi (siavashi@aut.ac.ir)
* Date: June, 2018
* Language: C++11
*/

#include <hip/hip_runtime.h>
#include <cstdlib>
#include <vector>
#include <chrono>
#include <iostream>>
#include <cmath>
#include <numeric>

// N = 2^27
#define N pow(2, 27)

using namespace std;

__global__ void reduce0(int *g_idata, int *g_odata) {
	extern __shared__ int sdata[];
	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[tid] = g_idata[i];
	__syncthreads();
	// do reduction in shared mem
	for(unsigned int s=1; s < blockDim.x; s *= 2) {
		if (tid % (2*s) == 0) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

int main(int argc, char *argv[]) {
	// initialize a vector of size N with 1
	vector<int> v(N, 1);
	// capture start time
	auto start_time = chrono::high_resolution_clock::now();
	// reduction
	auto sum = accumulate(begin(v), end(v), 0);
	// capture end time
	auto end_time = chrono::high_resolution_clock::now();
	// elapsed time in milliseconds
	auto duration = chrono::duration_cast<chrono::microseconds>(end_time - start_time);
	// print sum and elapsed time
	cout << "[-] Sum: " << sum << endl;
	cout << "[-] Duration: " << duration.count() << "ms" << endl;
	return EXIT_SUCCESS;
}
