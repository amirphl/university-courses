#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <cstdlib>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>

// C: array which contains result of compuation of histogram
// A: input array
__global__ void histo_kernel(int *A, int a_length, int *C, int c_length){
	__shared__ int* block_local_histo;
	if(threadIdx.x == 0){
		block_local_histo = (int *)malloc(c_length * sizeof(int));
	}
	__syncthreads();
	if(threadIdx.x < c_length) block_local_histo[threadIdx.x] = 0;
	__syncthreads();
	//printf("bug");
	int i = threadIdx.x + blockIdx.x * blockDim.x;	
	int stride = blockDim.x * gridDim.x;
    while (i < a_length) {
        atomicAdd(&(block_local_histo[A[i]]),1);
        i += stride;
    }
	__syncthreads();
	if (threadIdx.x < c_length) {
		atomicAdd(&(C[threadIdx.x]), block_local_histo[threadIdx.x] );
	}
}


void constantInit(int *data, int size, int range)
{
	for (int i = 0; i < size; ++i)
	{
		data[i] = rand() % range;
	}
}

void print(int* h_C, int c_length, int* h_A, int a_length){

	for(int i = 0; i < c_length; i++){
		printf("%d, ", h_C[i]);
	}
	printf("\n");
	for(int i = 0; i < a_length; i++){
		printf("%d, ", h_A[i]);
	}
}

/**
* Run a simple test of histogram calculation using CUDA
*/
int histogram_calc(int argc, char **argv, int n, int range)
{	
	// Allocate host memory for array A
	unsigned int size_A = n;
	unsigned int mem_size_A = sizeof(int)* size_A;
	int *h_A = (int *)malloc(mem_size_A);
	
	// Initialize host memory
	constantInit(h_A, size_A, range);
	
	// Allocate device memory
	int *d_A, *d_C;

	// Allocate host matrix C
	unsigned int mem_size_C = range * sizeof(int);
	int *h_C = (int *)malloc(mem_size_C);

	if (h_C == NULL)
	{
		fprintf(stderr, "Failed to allocate host matrix C!\n");
		exit(EXIT_FAILURE);
	}

	hipError_t error;

	error = hipMalloc((void **)&d_A, mem_size_A);

	if (error != hipSuccess)
	{
		printf("hipMalloc d_A returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void **)&d_C, mem_size_C);
	
	if (error != hipSuccess)
	{
		printf("hipMalloc d_C returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	// copy host memory to device
	error = hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);

	if (error != hipSuccess)
	{
		printf("hipMemcpy (d_A,h_A) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	// Setup execution parameters
	dim3 DimGrid(1,1,1);
	dim3 DimBlock(1024,1,1);
	
	// Create and start timer
	printf("Computing result using CUDA Kernel...\n");

	// Allocate CUDA events that we'll use for timing
	hipEvent_t start;
	error = hipEventCreate(&start);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	hipEvent_t stop;
	error = hipEventCreate(&stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Record the start event
	error = hipEventRecord(start, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Execute the kernel
	
	histo_kernel <<< DimGrid, DimBlock>>> (d_A, n, d_C, range);
	
	error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to launch kernel!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Record the stop event
	error = hipEventRecord(stop, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Wait for the stop event to complete
	error = hipEventSynchronize(stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	float msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);

	printf("Elapsed time in msec = %f\n", msecTotal);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Copy result from device to host
	error = hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

	if (error != hipSuccess)
	{
		printf("hipMemcpy (h_C,d_C) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}
	
	//print(h_C, range, h_A, n);
	

	// Clean up memory
	free(h_A);
	free(h_C);
	hipFree(d_A);
	hipFree(d_C);

	return EXIT_SUCCESS;

}


/**
* Program main
*/
int main(int argc, char **argv)
{
	printf("[Histogram Calculation Using CUDA] - Starting...\n");

	// By default, we use device 0
	int devID = 0;
	hipSetDevice(devID);

	hipError_t error;
	hipDeviceProp_t deviceProp;
	error = hipGetDevice(&devID);

	if (error != hipSuccess)
	{
		printf("hipGetDevice returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	}

	error = hipGetDeviceProperties(&deviceProp, devID);

	if (deviceProp.computeMode == hipComputeModeProhibited)
	{
		fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
		exit(EXIT_SUCCESS);
	}

	if (error != hipSuccess)
	{
		printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	}
	else
	{
		printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
	}

	// Size of input array
	int input_array_length = 1000000;
	// Size of input array
	int range = 5000;
	
	int result = histogram_calc(argc, argv, input_array_length, range);
	exit(result);
}
