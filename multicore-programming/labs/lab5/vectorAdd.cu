#include <iostream>
#include <stdlib.h>
#include "hip/hip_runtime.h"


void fillVector(int * v, size_t n);
void addVector(int * a, int *b, int *c, size_t n);
void printVector(int * v, size_t n);
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size, int multiplier);
hipError_t showIds();
__global__ void addKernel(int *c, const int *a, const int *b, int multiplier);
__global__ void addKernel_2(int *c, const int *a, const int *b, int multiplier);
__global__ void kernel_3(int *a,unsigned int *b,unsigned *c,unsigned int *d);

int main()
{	
	int multiplier = 100 * 1024 / 40; // 40 blocks for 5 SM
	int vectorSize = 100 * 1024 * 1024;
	
	int *a = (int *)malloc(vectorSize * sizeof(int));
	int *b = (int *)malloc(vectorSize * sizeof(int));
	int *c = (int *)malloc(vectorSize * sizeof(int));
	
	fillVector(a, vectorSize);
	fillVector(b, vectorSize);

	hipEvent_t start;
	hipEventCreate(&start);
	hipEventRecord(start, NULL);

	addWithCuda(c, a, b, vectorSize, multiplier);
	//showIds();

	hipEvent_t stop;
	hipEventCreate(&stop);
	hipEventRecord(stop, NULL);
	hipEventSynchronize(stop);
	
	float msecTotal = 0.0f;
	hipEventElapsedTime(&msecTotal, start, stop);
	printf ("time is %f miliseconds\n", msecTotal);
	//printVector(c, vectorSize);
	return EXIT_SUCCESS;
}

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size, int multiplier)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
	}
	
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
	}

	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
	}
	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
	}
	
	hipEvent_t start;
	hipEventCreate(&start);
	hipEventRecord(start, NULL);
	
	//addKernel <<< 1, 1024 >>>(dev_c, dev_a, dev_b);
	addKernel_2 <<< 40, 1024 >>>(dev_c, dev_a, dev_b,multiplier);
	
	hipEvent_t stop;
	hipEventCreate(&stop);
	hipEventRecord(stop, NULL);
	hipEventSynchronize(stop);
	
	float msecTotal = 0.0f;
	hipEventElapsedTime(&msecTotal, start, stop);
	printf ("time excluding arrays transmission is %f miliseconds\n", msecTotal);
	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}
	
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launchin addKernel!\n", cudaStatus);
	}
	
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
	}
	
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
	return cudaStatus;
}

hipError_t showIds(){
	
	int size = 2 * 64;
	
	int *dev_a = 0;
	unsigned int *dev_b = 0;
	unsigned *dev_c = 0;
	unsigned int *dev_d = 0;
	
	int *a = (int *)malloc(size * sizeof(int));
	unsigned int *b = (unsigned int *)malloc(size * sizeof(unsigned int));
	unsigned *c = (unsigned *)malloc(size * sizeof(unsigned));
	unsigned int *d = (unsigned int *)malloc(size * sizeof(unsigned int));;
	
	
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
	}
	
	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(long int));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(unsigned int));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(unsigned));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)&dev_d, size * sizeof(uint3));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
	}
	
	kernel_3 <<< 2, 64 >>>(dev_a, dev_b, dev_c, dev_d);
	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}
	
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launchin addKernel!\n", cudaStatus);
	}
	
	cudaStatus = hipMemcpy(a, dev_a, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
	}

	cudaStatus = hipMemcpy(b, dev_b, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
	}
	
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
	}
	
	cudaStatus = hipMemcpy(d, dev_d, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
	}
	
	for(int i = 0; i < size; i++){
		printf("Worker: %d -SMid: %d -Blockid: %d -Warpid: %d -Threadid: %d\n",i,a[i],b[i],c[i],d[i]);
	}
	
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipFree(dev_d);
	
	free(a);
	free(b);
	free(c);
	free(d);
	
	return cudaStatus;
}

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void addKernel_2(int *c, const int *a, const int *b, int multiplier)
{
	int thread_id = threadIdx.x + blockIdx.x;
	
	for(int j = thread_id * multiplier ; j < (thread_id + 1)* multiplier ; j++){
		c[j] = a[j] + b[j];
	}
}

//arrays a, b, c, d respectivly contain SMId, BlockId, WarpId and ThreadId
__global__ void kernel_3(int *a,unsigned int *b,unsigned *c,unsigned int *d)
{
	int t_id = (int) (threadIdx.x + blockIdx.x * blockDim.x);
	unsigned w_id; 
    asm volatile ("mov.u32 %0, %warpid;" : "=r"(w_id));
	long int smid;
	asm volatile("mov.u32 %0, %%smid;" : "=r"(smid));
	a[t_id] = smid;
	b[t_id] = blockIdx.x;
	c[t_id] = w_id;
	d[t_id] = threadIdx.x;
}

// Fills a vector with data
void fillVector(int * v, size_t n) {
	int i;
	for (i = 0; i < n; i++) {
		v[i] = i;
	}
}

// Adds two vectors
void addVector(int * a, int *b, int *c, size_t n) {
	int i;
	for (i = 0; i < n; i++) {
		c[i] = a[i] + b[i];
	}
}

// Prints a vector to the stdout.
void printVector(int * v, size_t n) {
	int i;
	printf("[-] Vector elements: ");
	for (i = 0; i < n; i++) {
		printf("%d, ", v[i]);
	}
	printf("\b\b  \n");
}
