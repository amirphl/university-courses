#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdlib.h>
#include <omp.h>
#include<iostream>
#define MAX_HISTORGRAM_NUMBER 10000
#define ARRAY_SIZE 81920000
#define tike 25600000


#define CHUNK_SIZE 1000
#define THREAD_COUNT 1024
#define BLOCK_COUNT 200
#define SCALER 20

__global__ void histogramKernelSingle(unsigned long long int *c, int *a)
{
	unsigned long long int worker =  blockIdx.x*blockDim.x + threadIdx.x;
	unsigned long long int start = worker * CHUNK_SIZE;
	unsigned long long int end = start + CHUNK_SIZE;
	for (int ex = 0; ex < SCALER; ex++)
		for (long long int i = start; i < end; i++)
		{
			if (i < ARRAY_SIZE / 4)
				atomicAdd(&c[a[i]], 1);
			else
				break;
		}

}


int main()
{
		hipSetDevice(0);
		//int* a_0 = (int*)malloc(sizeof(int)*ARRAY_SIZE/4);
		//int* a_1 = (int*)malloc(sizeof(int)*ARRAY_SIZE/4);
		//int* a_2 = (int*)malloc(sizeof(int)*ARRAY_SIZE/4);
		//int* a_3 = (int*)malloc(sizeof(int)*ARRAY_SIZE/4);
		int* a_0;
		int* a_1;
		int* a_2;
		int* a_3;
		
		
		hipHostMalloc((void**)&a_0, ARRAY_SIZE * sizeof(int) / 4);
		hipHostMalloc((void**)&a_1, ARRAY_SIZE * sizeof(int) / 4);
		hipHostMalloc((void**)&a_2, ARRAY_SIZE * sizeof(int) / 4);
		hipHostMalloc((void**)&a_3, ARRAY_SIZE * sizeof(int) / 4);
		
		
		int* b = (int*)malloc(sizeof(int)*ARRAY_SIZE/4);
		
		
		unsigned long long int *c_0 = (unsigned long long int*)malloc(MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int));
		unsigned long long int *c_1 = (unsigned long long int*)malloc(MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int));
		unsigned long long int *c_2 = (unsigned long long int*)malloc(MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int));
		unsigned long long int *c_3 = (unsigned long long int*)malloc(MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int));
		
		
		int *dev_a_0 = 0;
		int *dev_a_1 = 0;
		int *dev_a_2 = 0;
		int *dev_a_3 = 0;
		
		unsigned long long int *dev_c_0 = 0;
		unsigned long long int *dev_c_1 = 0;
		unsigned long long int *dev_c_2 = 0;
		unsigned long long int *dev_c_3 = 0;
		
		
		for (unsigned long long i = 0; i < ARRAY_SIZE/4 ;i++){
			//a_0[i] = rand() % MAX_HISTORGRAM_NUMBER;
			//a_1[i] = rand() % MAX_HISTORGRAM_NUMBER;
			//a_2[i] = rand() % MAX_HISTORGRAM_NUMBER;
			//a_3[i] = rand() % MAX_HISTORGRAM_NUMBER;
			b[i] = rand() % MAX_HISTORGRAM_NUMBER;
		}
		
		
		memcpy(a_0, b, ARRAY_SIZE * sizeof(int) / 4);
		memcpy(a_1, b, ARRAY_SIZE * sizeof(int) / 4);
		memcpy(a_2, b, ARRAY_SIZE * sizeof(int) / 4);
		memcpy(a_3, b, ARRAY_SIZE * sizeof(int) / 4);
		
		
		memset(c_0, 0, MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int));
		memset(c_1, 0, MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int));
		memset(c_2, 0, MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int));
		memset(c_3, 0, MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int));
		
		
		hipError_t cudaStatus;

    
		double start_time = omp_get_wtime();

		
		cudaStatus = hipMalloc((void**)&dev_a_0, ARRAY_SIZE * sizeof(int) / 4);
		cudaStatus = hipMalloc((void**)&dev_a_1, ARRAY_SIZE * sizeof(int) / 4);
		cudaStatus = hipMalloc((void**)&dev_a_2, ARRAY_SIZE * sizeof(int) / 4);
		cudaStatus = hipMalloc((void**)&dev_a_3, ARRAY_SIZE * sizeof(int) / 4);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc dev_a failed!");
			hipFree(dev_a_0);
			hipFree(dev_a_1);
			hipFree(dev_a_2);
			hipFree(dev_a_3);
			hipHostFree(a_0);
			hipHostFree(a_1);
			hipHostFree(a_2);
			hipHostFree(a_3);
			free(c_0);
			free(c_1);
			free(c_2);
			free(c_3);
			free(b);
			return 1;
		}
			
			
		cudaStatus = hipMalloc((void**)&dev_c_0, MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int));
		cudaStatus = hipMalloc((void**)&dev_c_1, MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int));
		cudaStatus = hipMalloc((void**)&dev_c_2, MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int));
		cudaStatus = hipMalloc((void**)&dev_c_3, MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc dev_c failed!");
			hipFree(dev_c_0);
			hipFree(dev_c_1);
			hipFree(dev_c_2);
			hipFree(dev_c_3);
			hipHostFree(a_0);
			hipHostFree(a_1);
			hipHostFree(a_2);
			hipHostFree(a_3);
			free(c_0);
			free(c_1);
			free(c_2);
			free(c_3);
			free(b);
			return 1;
		}
		
		
		hipStream_t stream_0;
		hipStream_t stream_1;
		hipStream_t stream_2;
		hipStream_t stream_3;
		hipStreamCreate(&stream_0);
		hipStreamCreate(&stream_1);
		hipStreamCreate(&stream_2);
		hipStreamCreate(&stream_3);
		
			
		hipMemcpyAsync(dev_a_0, a_0, ARRAY_SIZE * sizeof(int) / 4, hipMemcpyHostToDevice, stream_0);
		hipMemcpyAsync(dev_c_0, c_0, MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int), hipMemcpyHostToDevice, stream_0);
		histogramKernelSingle << <BLOCK_COUNT, THREAD_COUNT, 0, stream_0>> > (dev_c_0, dev_a_0);
		
		hipMemcpyAsync(dev_a_1, a_1, ARRAY_SIZE * sizeof(int) / 4, hipMemcpyHostToDevice, stream_1);
		hipMemcpyAsync(dev_c_1, c_1, MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int), hipMemcpyHostToDevice, stream_1);
		histogramKernelSingle << <BLOCK_COUNT, THREAD_COUNT, 0, stream_1>> > (dev_c_1, dev_a_1);
		
		hipMemcpyAsync(dev_a_2, a_2, ARRAY_SIZE * sizeof(int) / 4, hipMemcpyHostToDevice, stream_2);
		hipMemcpyAsync(dev_c_2, c_2, MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int), hipMemcpyHostToDevice, stream_2);
		histogramKernelSingle << <BLOCK_COUNT, THREAD_COUNT, 0, stream_2>> > (dev_c_2, dev_a_2);
		
		hipMemcpyAsync(dev_a_3, a_3, ARRAY_SIZE * sizeof(int) / 4, hipMemcpyHostToDevice, stream_3);
		hipMemcpyAsync(dev_c_3, c_3, MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int), hipMemcpyHostToDevice, stream_3);
		histogramKernelSingle << <BLOCK_COUNT, THREAD_COUNT, 0, stream_3>> > (dev_c_3, dev_a_3);
		
		hipMemcpyAsync(c_0, dev_c_0, MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int), hipMemcpyDeviceToHost, stream_0);
		hipMemcpyAsync(c_1, dev_c_1, MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int), hipMemcpyDeviceToHost, stream_1);
		hipMemcpyAsync(c_2, dev_c_2, MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int), hipMemcpyDeviceToHost, stream_2);
		hipMemcpyAsync(c_3, dev_c_3, MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int), hipMemcpyDeviceToHost, stream_3);
		
		
		hipStreamSynchronize(stream_0);
		hipStreamSynchronize(stream_1);
		hipStreamSynchronize(stream_2);
		hipStreamSynchronize(stream_3);
		
		
		hipStreamDestroy(stream_0);
		hipStreamDestroy(stream_1);
		hipStreamDestroy(stream_2);
		hipStreamDestroy(stream_3);
		
		
		hipDeviceSynchronize();
		
		
		double end_time = omp_get_wtime();
		std::cout << end_time - start_time;

		hipFree(dev_a_0);
		hipFree(dev_a_1);
		hipFree(dev_a_2);
		hipFree(dev_a_3);
		hipFree(dev_c_0);
		hipFree(dev_c_1);
		hipFree(dev_c_2);
		hipFree(dev_c_3);
		hipHostFree(a_0);
		hipHostFree(a_1);
		hipHostFree(a_2);
		hipHostFree(a_3);
		
	
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}
		
		unsigned long long int R = 0;
		
		for (int i = 0; i < MAX_HISTORGRAM_NUMBER; i++){
			R += c_0[i] + c_1[i] + c_2[i] + c_3[i];
		}
		
		printf("\n%lld		%d		%d", R/(SCALER) , ARRAY_SIZE, ARRAY_SIZE == R/(SCALER));
		
		free(c_0);
		free(c_1);
		free(c_2);
		free(c_3);
		free(b);
		
		printf("\nCHUNK_SIZE: %d\nTHREAD_COUNT: %d\nBLOCK_COUNT: %d\n" , CHUNK_SIZE, THREAD_COUNT,BLOCK_COUNT);
		return 0;
}