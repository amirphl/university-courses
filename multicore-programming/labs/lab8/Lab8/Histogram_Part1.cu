#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdlib.h>
#include <omp.h>
#include<iostream>
#define MAX_HISTORGRAM_NUMBER 10000
#define ARRAY_SIZE 102400000

#define CHUNK_SIZE 100
#define THREAD_COUNT 8
#define BLOCK_COUNT 128000
#define SCALER 20
hipError_t histogramWithCuda(int *a, unsigned long long int *c);

__global__ void histogramKernelSingle(unsigned long long int *c, int *a)
{
	unsigned long long int worker =  blockIdx.x*blockDim.x + threadIdx.x;
	unsigned long long int start = worker * CHUNK_SIZE;
	unsigned long long int end = start + CHUNK_SIZE;
	for (int ex = 0; ex < SCALER; ex++)
		for (long long int i = start; i < end; i++)
		{
			if (i < ARRAY_SIZE)
				atomicAdd(&c[a[i]], 1);
			else
			{

				break;
			}
		}

}
int main()
{
		int* a = (int*)malloc(sizeof(int)*ARRAY_SIZE);
		unsigned long long int* c = (unsigned long long int*)malloc(sizeof(unsigned long long int)*MAX_HISTORGRAM_NUMBER);
		for (unsigned long long i = 0; i < ARRAY_SIZE;i++)
			a[i] = rand() % MAX_HISTORGRAM_NUMBER;
		for (unsigned long long i = 0; i < MAX_HISTORGRAM_NUMBER; i++)
			c[i] = 0;

    // Add vectors in parallel.
		double start_time = omp_get_wtime();
		hipError_t cudaStatus=histogramWithCuda(a,c);
		double end_time = omp_get_wtime();
		std::cout << end_time - start_time;
   // = 
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	unsigned long long int R = 0;
	for (int i = 0; i < MAX_HISTORGRAM_NUMBER; i++)
	{
		R += c[i];
//		printf("%d	", c[i]);
	}
	printf("\nCORRECT:%ld	", R/(SCALER));
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t histogramWithCuda(int *a, unsigned long long int *c)
{
    int *dev_a = 0;
	unsigned long long int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, ARRAY_SIZE * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    // Launch a kernel on the GPU with one thread for each element.
	//// BLOCK CALCULATOR HERE
	

	////BLOCK CALCULATOR HERE

	//dim3 blockSize(2000,1,1);
		histogramKernelSingle << <BLOCK_COUNT, THREAD_COUNT>> > (dev_c, dev_a);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	
Error:
    hipFree(dev_c);
    hipFree(dev_a);
    return cudaStatus;
}
